#include <iostream>

int main() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        std::cout << "Device Number: " << i << std::endl;
        std::cout << "  Device name: " << prop.name << std::endl;
        std::cout << "  Total global memory: " << prop.totalGlobalMem << " bytes" << std::endl;
        std::cout << "  Compute capability: " << prop.major << "." << prop.minor << std::endl;
        std::cout << "  Total constant memory: " << prop.totalConstMem << " bytes" << std::endl;
        std::cout << "  Shared memory per block: " << prop.sharedMemPerBlock << " bytes" << std::endl;
        std::cout << "  Registers per block: " << prop.regsPerBlock << std::endl;
        std::cout << "  Warp size: " << prop.warpSize << std::endl;
        std::cout << "  Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
        std::cout << "  Max dimensions of a block (x, y, z): "
                  << prop.maxThreadsDim[0] << ", "
                  << prop.maxThreadsDim[1] << ", "
                  << prop.maxThreadsDim[2] << std::endl;
        std::cout << "  Max dimensions of a grid (x, y, z): "
                  << prop.maxGridSize[0] << ", "
                  << prop.maxGridSize[1] << ", "
                  << prop.maxGridSize[2] << std::endl;
        std::cout << "  Memory Clock Rate (KHz): " << prop.memoryClockRate << std::endl;
        std::cout << "  Memory Bus Width (bits): " << prop.memoryBusWidth << std::endl;
        std::cout << "  Peak Memory Bandwidth (GB/s): "
                  << 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6 << std::endl;
    }
    return 0;
}
