#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "../ops.h"
#define M 256  // Number of rows in A and C
#define K 512   // Number of columns in A and rows in B
#define N 256  // Number of columns in B and C
#define BLOCK_SIZE 16


// CPU matrix multiplication
void matmul_cpu(float *A, float *B, float *C, int m, int k, int n) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            float sum = 0.0f;
            for (int l = 0; l < k; l++) {
                sum += A[i * k + l] * B[l * n + j];
            }
            C[i * n + j] = sum;
        }
    }
}

// CUDA kernel for matrix multiplication
__global__ void matmul_gpu(float *A, float *B, float *C, int m, int k, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        float sum = 0.0f;
        for (int l = 0; l < k; l++) {
            sum += A[row * k + l] * B[l * n + col];
        }
        C[row * n + col] = sum;
    }
}

// Initialize matrix with random values
void init_matrix(float *mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        mat[i] = (float)rand() / RAND_MAX;
    }
}


int main() {
    float *h_A, *h_B, *h_C_cpu, *h_C_gpu;
    float *d_A, *d_B, *d_C;
    int size_A = M * K * sizeof(float);
    int size_B = K * N * sizeof(float);
    int size_C = M * N * sizeof(float);

    // Allocate host memory
    h_A = (float*)malloc(size_A);
    h_B = (float*)malloc(size_B);
    h_C_cpu = (float*)malloc(size_C);
    h_C_gpu = (float*)malloc(size_C);

    // Initialize matrices
    srand(time(NULL));
    init_matrix(h_A, M, K);
    init_matrix(h_B, K, N);

    // Allocate device memory
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    // Copy data to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(ceil(N / BLOCK_SIZE), ceil(M / BLOCK_SIZE));

    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 3; i++) {
        matmul_cpu(h_A, h_B, h_C_cpu, M, K, N);
        matmul_gpu<<<blocks, threads>>>(d_A, d_B, d_C, M, K, N);
        hipDeviceSynchronize();
    }

    printf("Benchmarking CPU implementation...\n");
    measure_exec_time(matmul_cpu, h_A, h_B, h_C_cpu, M, K, N);

    printf("Benchmarking GPU implementation...\n");
    measure_exec_time(matmul_gpu, d_A, d_B, d_C, M, K, N);

    // Free memory
    free(h_A);
    free(h_B);
    free(h_C_cpu);
    free(h_C_gpu);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}